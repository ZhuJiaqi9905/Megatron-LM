#include <iostream>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_fp16.h> // For FP16 data type
#include <chrono>

#define CUDA_CHECK(status) \
    if (status != hipSuccess) { \
        std::cerr << "CUDA Error: " << hipGetErrorString(status) << std::endl; \
        exit(EXIT_FAILURE); \
    }

#define CUBLAS_CHECK(status) \
    if (status != HIPBLAS_STATUS_SUCCESS) { \
        std::cerr << "cuBLAS Error: " << status << std::endl; \
        exit(EXIT_FAILURE); \
    }

// Kernel to initialize FP16 matrices
__global__ void init_matrix_fp16(__half* matrix, int size, __half value) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        matrix[idx] = value;
    }
}

int main() {
    // Matrix dimensions
    const int N = 4096*8; // Matrix size N x N
    const int repeat_count = 20; // Number of repetitions
    const __half alpha = __float2half(1.0f);
    const __half beta = __float2half(0.0f);

    // Calculate the number of floating-point operations
    long long flops_per_matrix = 2LL * N * N * N;
    long long total_flops = flops_per_matrix * repeat_count;

    // Allocate device memory for FP16 matrices
    __half *d_A, *d_B, *d_C;
    CUDA_CHECK(hipMalloc((void**)&d_A, N * N * sizeof(__half)));
    CUDA_CHECK(hipMalloc((void**)&d_B, N * N * sizeof(__half)));
    CUDA_CHECK(hipMalloc((void**)&d_C, N * N * sizeof(__half)));

    // Initialize matrices on device
    int threads_per_block = 256;
    int blocks = (N * N + threads_per_block - 1) / threads_per_block;
    init_matrix_fp16<<<blocks, threads_per_block>>>(d_A, N * N, __float2half(1.0f));
    init_matrix_fp16<<<blocks, threads_per_block>>>(d_B, N * N, __float2half(1.0f));
    init_matrix_fp16<<<blocks, threads_per_block>>>(d_C, N * N, __float2half(0.0f));
    CUDA_CHECK(hipDeviceSynchronize());

    // Create cuBLAS handle and enable Tensor Core
    hipblasHandle_t handle;
    CUBLAS_CHECK(hipblasCreate(&handle));
    CUBLAS_CHECK(hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH));

    // Measure time for repeated matrix multiplications
    auto start = std::chrono::high_resolution_clock::now();

    for (int i = 0; i < repeat_count; ++i) {
        CUBLAS_CHECK(hipblasGemmEx(handle, 
                                  HIPBLAS_OP_N, HIPBLAS_OP_N, 
                                  N, N, N, 
                                  &alpha, 
                                  d_A, HIP_R_16F, N, 
                                  d_B, HIP_R_16F, N, 
                                  &beta, 
                                  d_C, HIP_R_16F, N, 
                                  HIP_R_16F, // Compute type: FP16
                                  CUBLAS_GEMM_DEFAULT_TENSOR_OP));
    }

    CUDA_CHECK(hipDeviceSynchronize());

    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed = end - start;

    // Calculate performance in TFLOPS
    double tflops = (total_flops / elapsed.count()) / 1e12;

    // Print results
    std::cout << "Matrix size: " << N << " x " << N << std::endl;
    std::cout << "Repetitions: " << repeat_count << std::endl;
    std::cout << "Time elapsed: " << elapsed.count() << " seconds" << std::endl;
    std::cout << "Performance: " << tflops << " TFLOPS" << std::endl;

    // Cleanup
    CUBLAS_CHECK(hipblasDestroy(handle));
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_C));

    return 0;
}
